#include "maelstrom/storage/storage.h"
#include <hip/hip_runtime.h>

namespace maelstrom {

    std::any create_stream(storage s) {
        auto t = maelstrom::single_storage_of(s);
        switch(t) {
            case DEVICE:
            case MANAGED: {
                hipStream_t st;
                hipStreamCreate(&st);
                return st;
            }
            case PINNED:
            case HOST: {
                return std::any();
            }
        }

        throw std::invalid_argument("Invalid storage");
    }

    void destroy_stream(storage s, std::any stream) {
        hipStreamDestroy(
            std::any_cast<hipStream_t>(stream)
        );
    }

    std::any get_default_stream(storage s) { return (hipStream_t)hipStreamDefault; }

}