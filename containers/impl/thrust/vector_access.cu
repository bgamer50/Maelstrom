#include "maelstrom/containers/vector.h"

#include <vector>
#include <sstream>

#include <hip/hip_runtime.h>

namespace maelstrom {

    std::any vector::get(size_t i) {
        if(i >= this->filled_size) throw std::out_of_range("Attempted to get element out of bounds!");

        size_t data_size = maelstrom::size_of(this->dtype);
        std::vector<unsigned char> raw_value(data_size);
        void* ptr = raw_value.data();

        if(this->mem_type == maelstrom::storage::DEVICE) {
            hipMemcpy(
                ptr,
                static_cast<unsigned char*>(this->data_ptr) + (data_size * i),
                data_size,
                hipMemcpyDefault
            );
        } else {
            ptr = static_cast<unsigned char*>(this->data_ptr) + (data_size * i);
        }

        return this->dtype.deserialize(ptr);
    }

    void vector::erase(size_t i) {
        if(this->filled_size == 0) {
            throw std::out_of_range("Can't erase elements of an empty vector");
        }

        if(i >= this->filled_size) {
            std::stringstream sx;
            sx << "Element " << i << " is out of range for vector of size " << this->filled_size;
            throw std::out_of_range(sx.str());
        }

        if(i == 0 && this->filled_size == 1) {
            return this->clear();
        }

        auto data_size = maelstrom::size_of(this->dtype);

        auto erase_ptr = static_cast<unsigned char*>(this->data_ptr) + (data_size * (i+1));
        auto new_data = static_cast<unsigned char*>(this->alloc(this->reserved_size));

        this->copy(this->data_ptr, new_data, i-1);
        this->copy(erase_ptr, new_data + (data_size * i), this->filled_size - i - 1);

        this->dealloc(this->data_ptr);
        this->data_ptr = new_data;

        this->filled_size -= 1;
    }

    maelstrom::vector as_host_vector(maelstrom::vector& vec) {
        if(vec.get_mem_type() == maelstrom::storage::HOST) {
            return maelstrom::vector(vec, true);
        }

        return vec.to(maelstrom::storage::HOST);
    }

    maelstrom::vector as_device_vector(maelstrom::vector& vec) {
        if(vec.get_mem_type() == maelstrom::storage::DEVICE) {
            return maelstrom::vector(vec, true);
        }

        return vec.to(maelstrom::storage::DEVICE);
    }

    maelstrom::vector as_primitive_vector(maelstrom::vector& vec, bool view) {
        auto prim_dtype = maelstrom::dtype_from_prim_type(vec.get_dtype().prim_type);

        if(vec.size() == 0) {
            return maelstrom::vector(
                vec.get_mem_type(),
                vec.get_dtype()
            );
        }

        return maelstrom::vector(
            vec.get_mem_type(),
            prim_dtype,
            vec.data(),
            vec.size(),
            view
        );
    }

}