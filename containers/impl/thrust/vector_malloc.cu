#include "maelstrom/containers/vector.h"
#include "maelstrom/util/cuda_utils.cuh"

#include <hip/hip_runtime.h>

#include <sstream>
#include <iostream>

namespace maelstrom {

    void* maelstrom::vector::alloc(size_t N) {
        size_t dtype_size = maelstrom::size_of(this->dtype);

        // Calls the base allocator
        auto base_mem_type = maelstrom::single_storage_of(this->mem_type);

        switch(base_mem_type) {
            case HOST: {
                void* ptr;
                hipMallocManaged(&ptr, dtype_size * N);
                hipMemAdvise(ptr, dtype_size * N, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
                hipDeviceSynchronize();
                maelstrom::cuda::cudaCheckErrors("vector alloc host memory");
                return ptr;
            }
            case DEVICE: {
                void* ptr;
                hipMallocAsync(&ptr, dtype_size * N, std::any_cast<hipStream_t>(this->stream));
                hipDeviceSynchronize();
                maelstrom::cuda::cudaCheckErrors("vector alloc device memory");
                return ptr;
            }
            case MANAGED: {
                void* ptr;
                hipMallocManaged(&ptr, dtype_size * N);
                hipDeviceSynchronize();
                std::stringstream sx;
                sx << "vector alloc managed memory (" << this->name << ")";
                maelstrom::cuda::cudaCheckErrors(sx.str());
                return ptr;
            }
            case PINNED: {
                void* ptr;
                hipHostMalloc(&ptr, dtype_size * N);
                hipDeviceSynchronize();
                maelstrom::cuda::cudaCheckErrors("vector alloc pinned memory");
                return ptr;
            }
        }

        throw std::runtime_error("Invalid memory type provided to vector alloc()");
    }

    void maelstrom::vector::dealloc(void* ptr) {
        if(ptr == nullptr) throw std::invalid_argument("Cannot deallocate a null pointer");

        // Calls the base allocator
        auto base_mem_type = maelstrom::single_storage_of(this->mem_type);
        auto current_stream = std::any_cast<hipStream_t>(this->stream);

        switch(base_mem_type) {
            case HOST: {
                hipFree(ptr);
                hipDeviceSynchronize();
                std::stringstream sx;
                sx << "vector dealloc host-advised managed memory (" << this->name << ")";
                maelstrom::cuda::cudaCheckErrors(sx.str());
                return;
            }
            case MANAGED: {
                hipFree(ptr);
                hipDeviceSynchronize();
                std::stringstream sx;
                sx << "vector dealloc managed memory (" << this->name << ")";
                maelstrom::cuda::cudaCheckErrors(sx.str());
                return;
            }
            case DEVICE: {
                hipFreeAsync(ptr, current_stream);
                hipStreamSynchronize(current_stream);
                std::stringstream sx;
                sx << "vector dealloc device memory (" << this->name << ")";
                maelstrom::cuda::cudaCheckErrors(sx.str());
                return;
            }
            case PINNED: {
                hipHostFree(ptr);
                hipDeviceSynchronize();
                std::stringstream sx;
                sx << "vector dealloc pinned memory (" << this->name << ")";
                maelstrom::cuda::cudaCheckErrors(sx.str());
                return;
            }
        }

        throw std::runtime_error("Invalid memory type provided to vector dealloc");
    }

    // Copies from src (first arg) to dst (second arg) using hipMemcpy.
    void maelstrom::vector::copy(void* src, void* dst, size_t size) {
        if(src == dst) return;
        auto current_stream = std::any_cast<hipStream_t>(this->stream);

        hipMemcpyAsync(dst, src, maelstrom::size_of(this->dtype) * size, hipMemcpyDefault, current_stream);
        hipStreamSynchronize(current_stream);
        maelstrom::cuda::cudaCheckErrors("maelstrom vector copy");
    }

    void maelstrom::vector::pin() {
        if(!this->is_view() || this->mem_type != maelstrom::HOST) throw std::domain_error("Vector must be a host view to be pinned!");

        hipHostRegister(this->data_ptr, maelstrom::size_of(this->dtype) * this->local_size(), hipHostRegisterReadOnly);
        hipDeviceSynchronize();
        maelstrom::cuda::cudaCheckErrors("maelstrom vector pin");
    }

    void maelstrom::vector::unpin() {
        if(!this->is_view() || this->mem_type != maelstrom::HOST) throw std::domain_error("Vector must be a host view to be unpinned!");

        hipHostUnregister(this->data_ptr);
        hipDeviceSynchronize();
        maelstrom::cuda::cudaCheckErrors("maelstrom vector unpin");
    }

}